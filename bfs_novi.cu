#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <deque>
#include <iostream>
#include <time.h>

#define BLOCK_SIZE 1024
#define W_SIZE 32
#define INF 100000
#define WARPS 32
#define TILE_SIZE 64

 #define NUM_BANKS 16  
 #define LOG_NUM_BANKS 4  
 #define CONFLICT_FREE_OFFSET(n) ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))  
/*
__device__ int warpCull(int neighbor) {
	volatile __shared__ int scratch[WARPS][128];
	int hash = neighbor & 127;
	scratch[warp_id][hash] = neighbor;
	int retrieved = scratch[warp_id][hash];
	if(retrieved == neighbor) {
		scratch[warp_id][hash] = thread_id;
		if(scratch[warp_id][hash] != thread_id) {
			return 1;
		}
	}
	return 0;
}

__device__ int historyCull(int neighbor) {
	
}

/* Threads in a warp together copy their part of array - for INT
 *	- W_OFF - position of a thread in a warp (warp offset)
 *	- size	- size of array to be copied
 */
 /*
__device_ void memcpy_SIMD_int(int W_OFF, int size, int *dest, int *src) {
	for(int i = W_OFF, i < size; i += W_SIZE) {
		dest[i] = src[i];
	}
	
	__threadfence_block();
}
*/
__device__ void gatherWarp(int *queue, int queue_size, int *d_R, int *d_C) {
	volatile __shared__ int comm[WARPS][3];
	
	int lane_id = threadIdx.x % W_SIZE;
	int warp_id = threadIdx.x / W_SIZE;
	
	int node = 0, r = 0, r_end = 0;
	
	if(threadIdx.x < queue_size) {
		node = queue[threadIdx.x];
		r = d_R[node];
		r_end = d_R[node + 1];
	}
		
	while(__any(r_end - r > 0)) {
		
		if(r_end - r > 0) {
			comm[warp_id][0] = lane_id;
		}
		
		if(comm[warp_id][0] == lane_id) {
			comm[warp_id][1] = r;
			comm[warp_id][2] = r_end;
			r = r_end;
		}
		
		int r_gather = comm[warp_id][1] + lane_id;
		int r_gather_end = comm[warp_id][2];
		while(r_gather < r_gather_end) {
			volatile int neighbor = d_C[r_gather];
			r_gather += W_SIZE;
		}
	}
}


__device__ void prefix_sum(int *g_odata, int *g_idata, int n) {

	__shared__ int temp[BLOCK_SIZE]; 
	int thid = threadIdx.x;  
	int offset = 1;  
	
	int ai = thid;  
    int bi = thid + (n/2); 
	
    int bankOffsetA = CONFLICT_FREE_OFFSET(ai);  
    int bankOffsetB = CONFLICT_FREE_OFFSET(bi);  
    temp[ai + bankOffsetA] = g_idata[ai];  
    temp[bi + bankOffsetB] = g_idata[bi];
	for (int d = n>>1; d > 0; d >>= 1) {                   // build sum in place up the tree  
		__syncthreads();  
		if (thid < d) {         
			int ai = offset*(2*thid+1)-1;  
			int bi = offset*(2*thid+2)-1;  
			ai += CONFLICT_FREE_OFFSET(ai);  
			bi += CONFLICT_FREE_OFFSET(bi);
		   temp[bi] += temp[ai];  
		}  
		offset *= 2;  	
	
	}
	if(thid == 0) {
		temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0;
	}  
	
	for (int d = 1; d < n; d *= 2) {// traverse down tree & build scan  
		offset >>= 1;  
		__syncthreads();  
		if (thid < d) {
			int ai = offset*(2*thid+1)-1;  
			int bi = offset*(2*thid+2)-1;  
			ai += CONFLICT_FREE_OFFSET(ai);  
			bi += CONFLICT_FREE_OFFSET(bi);
			int t = temp[ai];  
			temp[ai] = temp[bi];  
			temp[bi] += t;   
		}  
	}  
	__syncthreads();

	g_odata[ai] = temp[ai + bankOffsetA];  
	g_odata[bi] = temp[bi + bankOffsetB];  

}

__device__ void prescan(int *g_odata, int *g_idata, int n)  {  
    __shared__ int temp[BLOCK_SIZE * 2];
    int thid = threadIdx.x;  
    int offset = 1;  

    temp[2*thid] = g_idata[2*thid];   
    temp[2*thid+1] = g_idata[2*thid+1];  
  	
    for (int d = n>>1; d > 0; d >>= 1) {   
		__syncthreads();  
		if (thid < d) {  
			int ai = offset*(2*thid+1)-1;  
			int bi = offset*(2*thid+2)-1;  
			temp[bi] += temp[ai];  
		}  
		offset *= 2;  
	}
         
    if (thid == 0) {
		temp[n - 1] = 0;
	}
            
    for (int d = 1; d < n; d *= 2) {  
         offset >>= 1;  
         __syncthreads();  
         if (thid < d) {  
			int ai = offset*(2*thid+1)-1;  
			int bi = offset*(2*thid+2)-1;  
			int t = temp[ai];  
			temp[ai] = temp[bi];  
			temp[bi] += t;   
         }  
    }  
     __syncthreads();  
	
    g_odata[2*thid] = temp[2*thid];
    g_odata[2*thid+1] = temp[2*thid+1];  

}  
	


__device__ void gatherCTA(int *queue, int *out_queue, int queue_size, int level, int *dist, int *offset, int *d_R, int *d_C, int *broj, int *obrada) {
	volatile __shared__ int comm[3];
	__shared__ int scan_input[BLOCK_SIZE * 2];
	__shared__ int scan_output[BLOCK_SIZE * 2];
	__shared__ int susjedi;
	int id = threadIdx.x;
	int node = 0, r = 0, r_end = 0;
	
	if(threadIdx.x < queue_size) {
		node = queue[id];
		r = d_R[node];
		r_end = d_R[node + 1];
	}
	
	while(true) {
		if(id == 0) {
			comm[0] = -1;
		}
		
		__syncthreads();
		
		if(r_end - r > 0) {
			comm[0] = id;
		}
		
		__syncthreads();
		
		if(comm[0] == -1) break;
		
		if(comm[0] == id) {
			//printf("inspekcija %d %d %d\n", level, node, r_end - r);
			comm[1] = r;
			comm[2] = r_end;
			r = r_end;
			susjedi = 0;
		}
		
		__syncthreads();
		
		int r_gather = comm[1] + id;
		int r_gather_end = comm[2];
		__shared__ int cta_offset;
		__shared__ int done;
		
		for(int i = 0; true; ++i) {
	
			if(id == 0) done = 1;
			__syncthreads();
			
			volatile int neighbor;
			scan_input[id] = 0;
			
			if(r_gather < r_gather_end) {
				done = 0;
				atomicAdd(&susjedi, 1);
				neighbor = d_C[r_gather];
				if(dist[neighbor] == -1) {
					dist[neighbor] = level;
					scan_input[id] = 1;
					atomicAdd(broj, 1);
					//printf("| %d %d |", queue[comm[0]], neighbor);
				}
			}
			__syncthreads();

			if(done == 1) break;
			
			if(threadIdx.x < BLOCK_SIZE) {
				prescan(scan_output, scan_input, BLOCK_SIZE);
			}
			
			__syncthreads();
			
			if(threadIdx.x == 0) {
				cta_offset = atomicAdd(offset, scan_output[BLOCK_SIZE-1] + scan_input[BLOCK_SIZE-1]);
			}

			__syncthreads();

			if(r_gather < r_gather_end && scan_input[id] == 1) {
				out_queue[scan_output[id] + cta_offset] = neighbor;
			}

			r_gather += BLOCK_SIZE;
		}
	}
}


__global__ void BFS(int *d_R, int *d_C, int numberOfNodes, int *queue, int *out_queue, int *d_dist, int *offset, int level, int *broj, int *obrada) {
	//if(threadIdx.x==0) printf("%d %d\n", blockIdx.x, level);
	int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
	
	__shared__ int neighbors[TILE_SIZE];
	int chunk_size = min(TILE_SIZE, numberOfNodes - TILE_SIZE * blockIdx.x);
	
	//if(threadIdx.x==0) printf("%d %d %d\n", blockIdx.x, level, chunk_size);

	if(threadIdx.x < chunk_size) {
		neighbors[threadIdx.x] = queue[blockIdx.x * TILE_SIZE + threadIdx.x];
	//	printf("tadadam %d %d %d --> %d\n", blockIdx.x, threadIdx.x, level, neighbors[threadIdx.x]);
	}
	//if(threadIdx.x==0) printf("chunk %d %d %d\n", level, blockIdx.x, chunk_size);
	__syncthreads();
	gatherCTA(neighbors, out_queue, chunk_size, level, d_dist, offset, d_R, d_C, broj, obrada);
	
//	if(threadIdx.x==0) printf("gotovo %d %d\n", blockIdx.x, level);

}

///////////////////////////////////////////////////
//	MAIN PROGRAM
///////////////////////////////////////////////////

int main(void) {

	const int ZERO = 0;

	int numberOfNodes;
	int numberOfEdges;
	int startingNode;
		
	scanf("%d", &numberOfNodes);
	scanf("%d", &numberOfEdges);
	scanf("%d", &startingNode);
	
	int *h_C = (int*) malloc(numberOfEdges * sizeof(int)) ;
	int *h_R = (int*) malloc((numberOfNodes + 1) * sizeof(int));
	int *h_dist = (int*) malloc(numberOfNodes * sizeof(int));
	int *h_q2 = (int*) malloc(numberOfNodes * sizeof(int));

	for(int i = 0; i < numberOfEdges; ++i) {
		scanf("%d", &h_C[i] );
	}

	for(int i = 0; i < numberOfNodes + 1; ++i) {
		scanf("%d", &h_R[i]);
		if(i < numberOfNodes) {
			h_dist[i] = (i == startingNode) ? 0 : -1;
		}
	}
	
	
	int *d_C;
	hipMalloc((void**) &d_C, numberOfEdges * sizeof(int));
	hipMemcpy(d_C, h_C, numberOfEdges * sizeof(int), hipMemcpyHostToDevice);

	int *d_R;
	hipMalloc((void**) &d_R, (numberOfNodes + 1) * sizeof(int));
	hipMemcpy(d_R, h_R, (numberOfNodes + 1) * sizeof(int), hipMemcpyHostToDevice);

	int *d_dist;
	hipMalloc((void**) &d_dist, numberOfNodes * sizeof(int));
	hipMemcpy(d_dist, h_dist, numberOfNodes * sizeof(int), hipMemcpyHostToDevice);

	int *d_q1;
	hipMalloc((void**) &d_q1, numberOfNodes * sizeof(int));
	hipMemcpy( &d_q1[0], &startingNode, sizeof(int), hipMemcpyHostToDevice);
 
	int *d_q2;
	hipMalloc((void**) &d_q2, numberOfNodes * sizeof(int));

	int *offset;
	hipMalloc((void**) &offset, sizeof(int));
	hipMemcpy(offset, &ZERO, sizeof(int), hipMemcpyHostToDevice);
	
	int *broj;
	hipMalloc((void**) &broj, sizeof(int));
	hipMemcpy(broj, &ZERO, sizeof(int), hipMemcpyHostToDevice);
	
	int *obrada;
	hipMalloc((void**) &obrada, sizeof(int));
	hipMemcpy(obrada, &ZERO, sizeof(int), hipMemcpyHostToDevice);
	
	clock_t startTime = clock();
	
	for(int level = 1;; ++level) {

		int queueSize;
		
		if(level == 1) {
			queueSize = 1;
		} else {
			hipMemcpy(&queueSize, offset, sizeof(int), hipMemcpyDeviceToHost);
		}
		hipMemcpy(offset, &ZERO, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(broj, &ZERO, sizeof(int), hipMemcpyHostToDevice);
		
		if(queueSize == 0 || level > 20) {
			break;
		}
		
		int numberOfBlocks = (int)ceil((double)queueSize / (double)TILE_SIZE);
		
		//printf("size %d blocks %d\n", queueSize, numberOfBlocks);


		BFS<<<numberOfBlocks, BLOCK_SIZE>>>(d_R, d_C, queueSize, d_q1, d_q2, d_dist, offset, level, broj, obrada);

		hipDeviceSynchronize();
		hipMemcpy(&queueSize, offset, sizeof(int), hipMemcpyDeviceToHost);
		
		int h_broj;
		hipMemcpy(&h_broj, broj, sizeof(int), hipMemcpyDeviceToHost);
		printf("\nvelicine %d %d %d\n", level, h_broj, queueSize);
		
		hipMemcpy(h_q2, d_q2, queueSize * sizeof(int), hipMemcpyDeviceToHost);
		printf("\n---------------\n");
		for(int j = 0; j < queueSize; ++j) printf("%d x ", h_q2[j]);
		printf("\n---------------\n");
		fflush(stdout);
		int *tmp = d_q1;
		d_q1 = d_q2;
		d_q2 = tmp;
	}
	
	clock_t endTime = clock();
	printf("Execution time: %lf\n", (double)(endTime - startTime) / CLOCKS_PER_SEC);

	hipMemcpy(h_dist, d_dist, numberOfNodes * sizeof(int), hipMemcpyDeviceToHost);

	FILE *out = fopen("results_par.txt", "w");
	
	for(int i = 0; i < numberOfNodes; ++i) {
		fprintf(out, "%d\n", h_dist[i]);
	}

	fclose(out);

	hipFree(d_C);
	hipFree(d_R);
	hipFree(d_dist);
	hipFree(offset);
	hipFree(d_q1);
	hipFree(d_q2);
	free(h_C);
	free(h_R);
	free(h_dist);
	
	return 0;
}